#include "hip/hip_runtime.h"
#define EIGEN_NO_CUDA

#include <iostream>

#include <hip/hip_runtime.h>

#include "raymarcher/raymarcher.h"
#include "shader/shader.h"
#include "utils/rgba.cuh"
#include "../kernel/render.cuh"
#include "../kernel/renderdata.cuh"
#include "../kernel/cudautils.cuh"
#include "../kernel/distance.cuh"


#include <cuda_gl_interop.h>

Raymarcher::Raymarcher(std::unique_ptr<Window> w, const Scene& s, GLuint p) : window(std::move(w)), scene(s)  {

    //register PBO
    pbo = p;
    //last arg says that we intend on overwriting the contexts of the pbo     
    gpuErrorCheck( hipGraphicsGLRegisterBuffer(&cudaPboResource, pbo, cudaGraphicsMapFlagsWriteDiscard) );


    int width = scene.c_width, height = scene.c_height;
    float distToViewPlane = 0.1f, aspectRatio = scene.getCamera().getAspectRatio(width,height);
    float heightAngle = scene.getCamera().getHeightAngle();

    
    Eigen::Matrix4f inverseViewMatrix = scene.getCamera().getViewMatrix().inverse();
    float viewPlaneHeight = 2.f * distToViewPlane * std::tan(.5f*float(heightAngle));
    float viewPlaneWidth = viewPlaneHeight * aspectRatio;

    //map deviceImage to point to the PBO
    void* devPtr;
    size_t numBytes;
    gpuErrorCheck( hipGraphicsMapResources(1, &cudaPboResource, 0) );
    gpuErrorCheck( hipGraphicsResourceGetMappedPointer(&devPtr,&numBytes,cudaPboResource) );
    deviceImageData = (RGBA*)devPtr;

    //allocate GPU shapes on the device
    allocateDeviceRenderData();

    // allocate inverse view matrix on the device
    mat4 hostInverseViewMat = mat4(inverseViewMatrix.data());
    gpuErrorCheck( hipMalloc(&deviceInverseViewMat, sizeof(mat4)) );
    gpuErrorCheck( hipMemcpy(deviceInverseViewMat, &hostInverseViewMat, sizeof(mat4), hipMemcpyHostToDevice) );

    // allocate constants on the device
    gpuErrorCheck( hipMalloc(&deviceWidth, sizeof(int)) );
    gpuErrorCheck( hipMalloc(&deviceHeight, sizeof(int)) );
    gpuErrorCheck( hipMalloc(&deviceViewPlaneHeight, sizeof(float)) );
    gpuErrorCheck( hipMalloc(&deviceViewPlaneWidth, sizeof(float)) );

    gpuErrorCheck( hipMemcpy(deviceWidth, &width, sizeof(int), hipMemcpyHostToDevice) );
    gpuErrorCheck( hipMemcpy(deviceHeight, &height, sizeof(int), hipMemcpyHostToDevice) );
    gpuErrorCheck( hipMemcpy(deviceViewPlaneWidth, &viewPlaneWidth, sizeof(float), hipMemcpyHostToDevice) );
    gpuErrorCheck( hipMemcpy(deviceViewPlaneHeight, &viewPlaneHeight, sizeof(float), hipMemcpyHostToDevice) );


    
}

Raymarcher::~Raymarcher() {
    gpuErrorCheck( hipFree(deviceLights) );
    gpuErrorCheck( hipFree(deviceShapes) );
    gpuErrorCheck( hipFree(deviceRenderData) );
    gpuErrorCheck( hipFree(deviceInverseViewMat) );
    gpuErrorCheck( hipFree(deviceWidth) );
    gpuErrorCheck( hipFree(deviceHeight) );

    gpuErrorCheck( hipFree(deviceViewPlaneWidth) );
    gpuErrorCheck( hipFree(deviceViewPlaneHeight) );
    gpuErrorCheck( hipGraphicsUnmapResources(1, &cudaPboResource, 0) );
    gpuErrorCheck( hipGraphicsUnregisterResource(cudaPboResource) );
    std::cout << "Raymarcher Cleaned Up!" << std::endl;  
}

void Raymarcher::allocateDeviceRenderData() {
    GPURenderData hostRenderData;

    //global data
    hostRenderData.globalData = GPUSceneGlobalData(scene.globalData);

    //camera data
    hostRenderData.cameraData.pos = vec4(scene.cameraData.pos.data());
    hostRenderData.cameraData.look = vec4(scene.cameraData.look.data());
    hostRenderData.cameraData.up = vec4(scene.cameraData.up.data());
    hostRenderData.cameraData.heightAngle = scene.cameraData.heightAngle;
    hostRenderData.cameraData.aperture = scene.cameraData.aperture;
    hostRenderData.cameraData.focalLength = scene.cameraData.focalLength;

    //array sizes
    hostRenderData.numLights = scene.metaData.lights.size();
    hostRenderData.numShapes = scene.metaData.shapes.size();

    GPUSceneLightData* hostLights = new GPUSceneLightData[hostRenderData.numLights];
    GPURenderShapeData* hostShapes = new GPURenderShapeData[hostRenderData.numShapes];


    //copy lights
    for (int i = 0; i < hostRenderData.numLights; ++i) {
        const SceneLightData& cpuLight = scene.metaData.lights[i];
        hostLights[i].id = cpuLight.id;
        hostLights[i].type = static_cast<GPULightType>(cpuLight.type);
        hostLights[i].color = vec4(cpuLight.color.data());
        hostLights[i].function = vec3(cpuLight.function.data());
        hostLights[i].pos = vec4(cpuLight.pos.data());
        hostLights[i].dir = vec4(cpuLight.dir.data());
        hostLights[i].penumbra = cpuLight.penumbra;
        hostLights[i].angle = cpuLight.angle;
    }


    //copy shapes
    for (int i = 0; i < hostRenderData.numShapes; ++i) {
        const RenderShapeData cpuShape = scene.metaData.shapes[i];
        const SceneMaterial& cpuMaterial = cpuShape.primitive.material;
        
        GPUSceneMaterial gpuMaterial(
        vec4(cpuMaterial.cAmbient.data()),
        vec4(cpuMaterial.cDiffuse.data()),
        vec4(cpuMaterial.cSpecular.data()),
        cpuMaterial.shininess,
        vec4(cpuMaterial.cReflective.data()),
        vec4(cpuMaterial.cTransparent.data()),
        cpuMaterial.ior);

        GPUScenePrimitive gpuPrimitive(
            static_cast<GPUPrimitiveType>(cpuShape.primitive.type),
            gpuMaterial
        );

        printf("CPU primitive type: %d\n", cpuShape.primitive.type);
        printf("GPU primitive type: %d\n", gpuPrimitive.type);

        

        // set shape data
        Eigen::Matrix4f ctm = cpuShape.ctm;
        // std::cout << "Eigen CTM:\n" << ctm << std::endl;
        Eigen::Matrix3f upperBlock = ctm.block<3,3>(0,0);
        // std::cout << "Upper 3x3 block:\n" << upperBlock << std::endl;
        Eigen::Matrix3f inverseTransposeCtm = upperBlock.inverse().transpose();
        // std::cout << "IVT3:\n" << inverseTransposeCtm << std::endl;

        mat4 deviceCtm = mat4(ctm.data());
        mat4 deviceInverseCtm = mat4(cpuShape.inverseCtm.data());
        mat3 deviceIVT3 = mat3(inverseTransposeCtm.data());

        // print(deviceCtm, "DEVICE CTM BEFORE SHAPE DATA constructor: ");
        
        hostShapes[i] = GPURenderShapeData(
            gpuPrimitive,
            deviceCtm,
            deviceInverseCtm,
            deviceIVT3
        );


    }

    //allocate device shapes and lights
    gpuErrorCheck( hipMalloc(&deviceLights, hostRenderData.numLights * sizeof(GPUSceneLightData)) );
    gpuErrorCheck( hipMalloc(&deviceShapes, hostRenderData.numShapes * sizeof(GPURenderShapeData)) );
    gpuErrorCheck( hipMemcpy(deviceLights, hostLights, 
           hostRenderData.numLights * sizeof(GPUSceneLightData), 
           hipMemcpyHostToDevice) );
    gpuErrorCheck( hipMemcpy(deviceShapes, hostShapes, 
            hostRenderData.numShapes * sizeof(GPURenderShapeData), 
            hipMemcpyHostToDevice) );

    hostRenderData.lights = deviceLights;
    hostRenderData.shapes = deviceShapes;

    //allocate device GPURenderData
    gpuErrorCheck( hipMalloc(&deviceRenderData, sizeof(GPURenderData)) );
    gpuErrorCheck( hipMemcpy(deviceRenderData, &hostRenderData, sizeof(GPURenderData), hipMemcpyHostToDevice) );

    //free host arrays
    delete[] hostLights;
    delete[] hostShapes;
}

void Raymarcher::run() {

    if (glfwGetCurrentContext() == nullptr) {
        std::cerr << "Error: No OpenGL context" << std::endl;
        return;
    }

    while(!(*window).shouldClose()) {
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        glEnable(GL_DEPTH_TEST);

        if (shaderProgram == 0) {
            std::cerr << "Error: shaderProgram is 0" << std::endl;
            break;
        }

        //TODO: add some conditional logic (only when event handlers receive something)
        render();


        glUseProgram(shaderProgram);
        GET_GL_ERROR("After glUseProgram");

        glActiveTexture(GL_TEXTURE0);
        glBindTexture(GL_TEXTURE_2D, texture);
        GLint texLoc = glGetUniformLocation(shaderProgram, "ourTexture");
        if (texLoc == -1) {
            std::cout << "Warning: Could not find texture uniform" << std::endl;
        }
        glUniform1i(texLoc, 0);

        glBindVertexArray(quad.vao);
        GET_GL_ERROR("After glBindVertexArray");

        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
        GET_GL_ERROR("After glDrawElements");

        glfwSwapBuffers(window->glWindow);
        glfwPollEvents();

        GET_GL_ERROR("Loop() ERROR\n");
    }
}

void Raymarcher::render() {

    int width = scene.c_width, height = scene.c_height;
    dim3 blockSize(16,16);
    dim3 gridSize(
        (width + blockSize.x - 1) / blockSize.x,
        (height + blockSize.y - 1) / blockSize.y
    ); // number of blocks

    renderKernel<<<gridSize,blockSize>>>(
        deviceImageData,
        deviceRenderData,
        deviceInverseViewMat,
        deviceWidth,
        deviceHeight,
        deviceViewPlaneWidth,
        deviceViewPlaneHeight
    );
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
    gpuErrorCheck( hipDeviceSynchronize());

    
    // update texture from PBO
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBindTexture(GL_TEXTURE_2D, texture);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

}
